#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <algorithm>
#include <ctime>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/stat.h>
//#include "math_functions.h"

#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/flann/miniflann.hpp"

//#define cimg_use_jpeg


#define SIZE 10

#define db(a) cout << #a << " = " << a << endl
#define db2(a, b) cout << #a << " = " << a << " " << #b << " = " << b << endl

using namespace std;

inline bool existe(const std::string& name) {
	struct stat buffer;
	return (stat(name.c_str(), &buffer) == 0);
}

__device__ void convolution(int coordinate, float* d_arr, float* d_result, int width, int len, int* mask, int* dir, int* pos)
{
	float c = 0;
	for (int ii = 0; ii < 3; ii++)
	{
		for (int jj = 0; jj < 3; jj++)
		{
			int x = coordinate + width * dir[ii * 3 + jj] + pos[jj];
			if (x >= 0 && x < len)
				c += d_arr[x] * mask[ii * 3 + jj];
		}
	}
	/*printf("abs(cx) + abs(cy) = %f\n", abs(cx) + abs(cy));
	printf("sqrt(pow(cx, 2) + pow(cy, 2)) = %f\n", sqrt(pow(cx, 2.f) + pow(cy, 2.f)));*/
	d_result[coordinate] = c;
}

__global__ void deviceComputeGaussian(float* d_arr, float* d_result, int width, int len, int* mask, int* dir, int* pos) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x == 0 && y < 10)
		printf("[%d %d]\n", x, y);
	if (x * width + y < len)
		convolution(x * width + y, d_arr, d_result, width, len, mask, dir, pos);
}


void performGPU(string filename)
{
	cv::Mat imagen = cv::imread(filename, cv::IMREAD_GRAYSCALE);
	cv::Mat imagen_output(imagen.rows, imagen.cols, CV_8UC1);

	std::clock_t startt = clock();
	double duration;


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// depth, numColors, initialize
	int WIDTH = imagen.cols;
	int HEIGHT = imagen.rows;
	float *arr, *result;
	float *d_arr, *d_result;

	arr = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
	result = (float*)malloc(WIDTH * HEIGHT * sizeof(float));


	hipMalloc((void**)&d_arr, WIDTH * HEIGHT * sizeof(float));
	hipMalloc((void**)&d_result, WIDTH * HEIGHT * sizeof(float));

	int pos[3] = { -1, 0, 1 };
	int dir[9] = { -1, -1, -1, 0, 0, 0, 1, 1, 1 };
	int mask[9] = {
		1. / 9, 1. / 9, 1. / 9,
		1. / 9, 1. / 9, 1. / 9,
		1. / 9, 1. / 9, 1. / 9
	};

	int* d_pos, *d_dir, *d_mask;

	hipMalloc((void**)&d_pos, 3 * sizeof(int));
	hipMalloc((void**)&d_dir, 9 * sizeof(int));
	hipMalloc((void**)&d_mask, 9 * sizeof(int));


	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < HEIGHT; j++) {
			arr[i * WIDTH + j] = imagen.at<uchar>(j, i);
			result[i * WIDTH + j] = 0;
		}
	}

	hipMemcpy(d_arr, arr, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_result, result, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);

	dim3 BLOCKS = dim3(1024, 1024);
	dim3 THREADS = dim3(4, 4);

	/*dim3 BLOCKS(2, 2);
	dim3 THREADS(2, 2);*/



	hipMemcpy(d_pos, pos, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dir, dir, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_mask, mask, 9 * sizeof(float), hipMemcpyHostToDevice);


	hipEventRecord(start);
	deviceComputeGaussian << < BLOCKS, THREADS >> > (d_arr, d_result, WIDTH, WIDTH * HEIGHT, d_mask, d_dir, d_pos);
	hipMemcpy(result, d_result, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < HEIGHT; j++) {
			imagen_output.at<uchar>(j, i) = result[i * WIDTH + j];
		}
	}

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	duration = (std::clock() - startt) / (double)CLOCKS_PER_SEC;

	printf("GPU for image %s takes %.2f seconds\n", filename.c_str(), duration);
	printf("hipEventElapsedTime = %.2f ms\n", milliseconds);

	free(arr);


	hipFree(d_arr);
	hipFree(d_result);

	hipFree(d_pos);
	hipFree(d_dir);
	hipFree(d_result);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	//ac� mostramos la imagen resultante
	cv::imshow("Desenfoque Gaussiano", imagen_output);
	cv::waitKey(0);
}


int main(int argc, char** argv) {
	puts("hola :)");
	performGPU("lena.jpg");
	return 0;
}

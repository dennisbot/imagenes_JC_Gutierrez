#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <algorithm>
#include <ctime>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/stat.h>
//#include "math_functions.h"

#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/flann/miniflann.hpp"

//#define cimg_use_jpeg


#define SIZE 10

#define db(a) cout << #a << " = " << a << endl
#define db2(a, b) cout << #a << " = " << a << " " << #b << " = " << b << endl

using namespace std;

inline bool existe(const std::string& name) {
	struct stat buffer;
	return (stat(name.c_str(), &buffer) == 0);
}

__device__ void convolution(int coordinate, float* d_arr, float* gradient, int width, int len, int* mask, int* dir, int* pos)
{
	float c = 0;
	for (int ii = 0; ii < 3; ii++)
	{
		for (int jj = 0; jj < 3; jj++)
		{
			int x = coordinate + width * dir[ii * 3 + jj] + pos[jj];
			if (x >= 0 && x < len)
				c += d_arr[x] * mask[ii * 3 + jj];
		}
	}
	gradient[coordinate] = c;
}

__global__ void deviceComputeGradient(float* d_arr, float* gradient, int width, int len, int* mask, int* dir, int* pos) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x * width + y < len)
		convolution(x * width + y, d_arr, gradient, width, len, mask, dir, pos);
}

__global__ void deviceMerge(float* xGradient, float* yGradient, float* target, int width, int len) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int index = x * width + y;
	if (index < len)
		target[index] = abs(xGradient[index]) + abs(yGradient[index]);
}

//segunda versi�n (un s�lo m�todo)
__device__ void convolution2(int coordinate, float* d_arr, float* gradient, float* d2_gradient, int width, int len, int* gx_mask, int* gy_mask, int* dir, int* pos)
{
	float cx = 0, cy = 0;
	for (int ii = 0; ii < 3; ii++)
	{
		for (int jj = 0; jj < 3; jj++)
		{
			int x = coordinate + width * dir[ii * 3 + jj] + pos[jj];
			if (x >= 0 && x < len) {
				cx += d_arr[x] * gx_mask[ii * 3 + jj];
				cy += d_arr[x] * gy_mask[ii * 3 + jj];
			}
		}
	}
	gradient[coordinate] = abs(cx) + abs(cy);
	d2_gradient[coordinate] = sqrt(pow(cx, 2.f) + pow(cy, 2.f));
	if (coordinate < 12) {
		printf("abs(cx) + abs(cy) = %f\n", abs(cx) + abs(cy));
		printf("sqrt(pow(cx, 2) + pow(cy, 2)) = %f\n", sqrt(pow(cx, 2.f) + pow(cy, 2.f)));
	}
	
}

__global__ void deviceComputeGradient2(float* d_arr, float* gradient, float* d2_gradient, int width, int len, int* gx_mask, int* gy_mask, int* dir, int* pos) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x == 0 && y < 10)
		printf("[%d %d]\n", x, y);
	if (x * width + y < len)
		convolution2(x * width + y, d_arr, gradient, d2_gradient, width, len, gx_mask, gy_mask, dir, pos);
}


void performGPU(string filename)
{
	cv::Mat imagen = cv::imread(filename, cv::IMREAD_GRAYSCALE);

	std::clock_t startt = clock();
	double duration;
	cv::Mat result(imagen.rows, imagen.cols, CV_8UC1);
	cv::Mat resultb(imagen.rows, imagen.cols, CV_8UC1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// depth, numColors, initialize
	int WIDTH = imagen.cols;
	int HEIGHT = imagen.rows;
	float *arr, *gradient, *bgradient;
	float *d_arr, *d_gradient, *d2_gradient;

	arr = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
	gradient = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
	bgradient = (float*)malloc(WIDTH * HEIGHT * sizeof(float));

	hipMalloc((void**)&d_arr, WIDTH * HEIGHT * sizeof(float));
	hipMalloc((void**)&d_gradient, WIDTH * HEIGHT * sizeof(float));
	hipMalloc((void**)&d2_gradient, WIDTH * HEIGHT * sizeof(float));

	for (int i = 0; i < WIDTH; i++)
		for (int j = 0; j < HEIGHT; j++) {
			arr[i * WIDTH + j] = imagen.at<uchar>(j, i);
			gradient[i * WIDTH + j] = 0;
			bgradient[i * WIDTH + j] = 0;
		}

	hipMemcpy(d_arr, arr, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_gradient, gradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d2_gradient, bgradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);

	dim3 BLOCKS = dim3(1024, 1024);
	dim3 THREADS = dim3(4, 4);

	/*dim3 BLOCKS(2, 2);
	dim3 THREADS(2, 2);*/

	int pos[3] = { -1, 0, 1 };
	int dir[9] = { -1, -1, -1, 0, 0, 0, 1, 1, 1 };
	int Gx[9] = {
		-1, 0, 1,
		-2, 0, 2,
		-1, 0, 1
	};
	int Gy[9] = {
		1, 2, 1,
		0, 0, 0,
		-1, -2, -1
	};
	int* d_pos, *d_dir, *d_Gx, *d_Gy;

	hipMalloc((void**)&d_pos, 3 * sizeof(int));
	hipMalloc((void**)&d_dir, 9 * sizeof(int));
	hipMalloc((void**)&d_Gx, 9 * sizeof(int));
	hipMalloc((void**)&d_Gy, 9 * sizeof(int));

	hipMemcpy(d_pos, pos, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dir, dir, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Gx, Gx, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Gy, Gy, 9 * sizeof(float), hipMemcpyHostToDevice);


	hipEventRecord(start);
	deviceComputeGradient2 << < BLOCKS, THREADS >> > (d_arr, d_gradient, d2_gradient, WIDTH, WIDTH * HEIGHT, d_Gx, d_Gy, d_dir, d_pos);
	hipMemcpy(gradient, d_gradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(bgradient, d2_gradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < HEIGHT; j++) {
			result.at<uchar>(j, i) = gradient[i * WIDTH + j];
			resultb.at<uchar>(j, i) = bgradient[i * WIDTH + j];
		}
	}


	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	duration = (std::clock() - startt) / (double)CLOCKS_PER_SEC;

	printf("GPU for image %s takes %.2f seconds\n", filename.c_str(), duration);
	printf("hipEventElapsedTime = %.2f ms\n", milliseconds);

	free(arr);


	hipFree(d_arr);
	hipFree(d_gradient);

	hipFree(d_pos);
	hipFree(d_dir);
	hipFree(d_Gx);
	hipFree(d_Gy);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	//ac� si quieres ponemos en xgradient, ygradient
	cv::imshow("detecci�n de bordes Lena ABS", result);
	cv::imshow("detecci�n de bordes Lena SQRT", resultb);
	cv::waitKey(0);
}


int main2(int argc, char** argv) {
	performGPU("lena.jpg");
	return 0;
}

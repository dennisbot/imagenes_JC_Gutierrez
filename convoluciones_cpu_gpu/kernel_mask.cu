#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <ctime>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/stat.h>
//#include "math_functions.h"

#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/flann/miniflann.hpp"

//#define cimg_use_jpeg

#define db(a) cout << #a << " = " << a << endl;

#define SIZE 10

#define db(a) cout << #a << " = " << a << endl
#define db2(a, b) cout << #a << " = " << a << " " << #b << " = " << b << endl

using namespace std;

inline bool existe(const std::string& name) {
	struct stat buffer;
	return (stat(name.c_str(), &buffer) == 0);
}

#define CUDA_CALL(cuda_function, ...)  { \
    hipError_t status = cuda_function(__VA_ARGS__); \
    cudaEnsureSuccess(status, #cuda_function, false, __FILE__, __LINE__); \
}

bool cudaEnsureSuccess(hipError_t status, const char* status_context_description,
	bool die_on_error, const char* filename, unsigned line_number) {
	if (status_context_description == NULL)
		status_context_description = "";
	if (status == hipSuccess) {
#if REPORT_CUDA_SUCCESS
		cerr << "Succeeded: " << status_context_description << std::endl << std::flush;
#endif
		return true;
	}
	const char* errorString = hipGetErrorString(status);
	cerr << "CUDA Error: ";
	if (status_context_description != NULL) {
		cerr << status_context_description << ": ";
	}
	if (errorString != NULL) {
		cerr << errorString;
	}
	else {
		cerr << "(Unknown CUDA status code " << status << ")";
	}
	if (filename != NULL) {

		cerr << filename << ":" << line_number;
	}

	cerr << std::endl;
	if (die_on_error) {
		exit(EXIT_FAILURE);
		// ... or cerr << "FATAL ERROR" << etc. etc.
	}
	return false;
}

__device__ void convolucion(int coordinate, float* d_arr, float* d_result, int width, int len, float* mask, float* dir, float* pos, int lado) {
	float c = 0;
	for (int ii = 0; ii < lado; ii++) {
		for (int jj = 0; jj < lado; jj++) {
			int x = coordinate + width * dir[ii * lado + jj] + pos[jj];
			/*if (x < 12)
				printf("d_arr[x] = %f\n", d_arr[x]);*/
			if (x >= 0 && x < len)
				c += d_arr[x] * mask[ii * lado + jj];
		}
	}
	d_result[coordinate] = c;
	//d_result[coordinate] = d_arr[coordinate];	
}

__global__ void deviceComputeResult(float* d_arr, float* d_result, int width, int len, float* mask, float* dir, float* pos, int lado) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x * width + y < len)
		convolucion(x * width + y, d_arr, d_result, width, len, mask, dir, pos, lado);
}


void ejecutarGPU(string filename)
{

	float *pmask, *pdir;
	int *ppos, lado;
	
	cv::Mat imagen = cv::imread(filename, cv::IMREAD_GRAYSCALE);
	double ratio = imagen.cols * 1. / imagen.rows;
	cv::Mat imagen_output(imagen.rows, imagen.cols, CV_8UC1);

	dim3 BLOCKS = dim3(32 * 4 + 1, 32 * 4 + 1);
	dim3 THREADS = dim3(32, 32);
	
	for (int sizes = 9; sizes <= 9; sizes++) {
		lado = 2 * sizes + 1;

		ppos = new int[lado];
		pmask = new float[lado * lado];
		pdir = new float[lado * lado];

		int dir_value = -lado / 2;

		for (int idx_pos = 0; idx_pos < lado; idx_pos++) ppos[idx_pos] = dir_value++;

		for (int idx = 0; idx < lado * lado; idx++) pmask[idx] = 1. / (lado * lado);

		dir_value = -lado / 2;

		for (int idx_dir = 0; idx_dir < lado * lado; idx_dir++) {
			if (idx_dir % lado == 0 && idx_dir != 0) dir_value++;
			pdir[idx_dir] = dir_value;
		}

		std::clock_t startt = clock();
		double duration;

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		// depth, numColors, initialize
		int WIDTH = imagen.cols;
		printf("WIDTH = %d\n", WIDTH);

		int HEIGHT = imagen.rows;
		printf("HEIGHT = %d\n", HEIGHT);

		float *arr, *result;
		float *d_arr, *d_result;

		arr = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
		result = (float*)malloc(WIDTH * HEIGHT * sizeof(float));

		hipMalloc((void**)&d_arr, WIDTH * HEIGHT * sizeof(float));
		hipMalloc((void**)&d_result, WIDTH * HEIGHT * sizeof(float));

			
		for (int i = 0; i < WIDTH; i++)
			for (int j = 0; j < HEIGHT; j++) {
				arr[i * WIDTH + j] = imagen.at<uchar>(j, i);
				result[i * WIDTH + j] = 0;
			}

		auto the_error = hipMemcpy(d_arr, arr, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
		if (the_error != hipSuccess) {
			fprintf(stderr, "hipMemcpyHostToDevice arr -> d_arr : hipMemcpy failed: %s\n",
				hipGetErrorString(the_error));
			return;
		}
		the_error = hipMemcpy(d_result, result, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
		if (the_error != hipSuccess) {
			fprintf(stderr, "hipMemcpyHostToDevice result -> d_result : hipMemcpy failed: %s\n",
				hipGetErrorString(the_error));
			return;
		}

		float *d_pos, *d_dir, *d_mask;

		hipMalloc((void**)&d_pos, lado * sizeof(float));
		hipMalloc((void**)&d_dir, lado * lado * sizeof(float));
		hipMalloc((void**)&d_mask, lado * lado * sizeof(float));


		hipMemcpy(d_pos, ppos, lado * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_dir, pdir, lado * lado * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_mask, pmask, lado * lado * sizeof(float), hipMemcpyHostToDevice);

		hipEventRecord(start);

		deviceComputeResult << < BLOCKS, THREADS >> >(d_arr, d_result, WIDTH, WIDTH * HEIGHT, d_mask, d_dir, d_pos, lado);

		std::string error = hipGetErrorString(hipPeekAtLastError());

		printf("hipGetErrorString(hipPeekAtLastError()) = %s\n", error);
		error = hipGetErrorString(hipDeviceSynchronize());
		printf("hipGetErrorString(hipDeviceSynchronize()) = %s\n", error);

		the_error = hipMemcpy(result, d_result, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);

		if (the_error != hipSuccess) {
			fprintf(stderr, "hipMemcpyDeviceToHost d_result -> result : hipMemcpy failed: %s\n",
				hipGetErrorString(the_error));
			return;
		}

		hipEventRecord(stop);
		hipEventSynchronize(stop);

		for (int i = 0; i < WIDTH; i++) {
			for (int j = 0; j < HEIGHT; j++) {
				imagen_output.at<uchar>(j, i) = result[i * WIDTH + j];
			}
		}

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		duration = (std::clock() - startt) / (double)CLOCKS_PER_SEC;

		printf("GPU for image %s takes %.2f seconds\n", filename.c_str(), duration);
		printf("imagen con m�scara de tamanio %d, hipEventElapsedTime = %.2f ms\n", lado, milliseconds / 1000);
		
		ofstream out("measurements", ios::out | ios::app);
		out << lado << "\t" << milliseconds / 1000 << endl;
		out.close();

		free(arr);
		free(result);
			
		free(ppos);
		free(pdir);
		free(pmask);

		hipFree(d_arr);
		hipFree(d_result);

		hipFree(d_pos);
		hipFree(d_dir);
		hipFree(d_mask);


		hipEventDestroy(start);
		hipEventDestroy(stop);

		//ac� vamos a realizar una comparaci�n

		//ratio = w / h
		double new_height = 700;
		double new_width = new_height * ratio;

		int fill = 2 - to_string(lado).size();
		cv::imwrite("output/Img_with_kernelSize_" + string(fill > 0 ? fill : 0, '0') + to_string(lado) + "_" + filename, imagen_output);
		/*cvNamedWindow("imagen original", CV_WINDOW_NORMAL);
		cvResizeWindow("imagen original", new_width, new_height);
		cv::imshow("imagen original", imagen);

		cvNamedWindow("imagen con filtro gausiano", CV_WINDOW_NORMAL);
		cvResizeWindow("imagen con filtro gausiano", new_width, new_height);
		cv::imshow("imagen con filtro gausiano", imagen_output);*/
		cv::waitKey(0);
	}
		
}


int main(int argc, char** argv) {
	//ejecutarGPU("lena.jpg");
	ejecutarGPU("4096x4096.jpg");
	
	return 0;
}

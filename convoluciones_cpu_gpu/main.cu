#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "helper.h"
#include "interpolacion.h"
#include "convolucion.h"

#include <thread>
#include <mutex>

#define db(a) cout << #a << " = " << a << endl;
#define db2(a,b) cout << #a << "= " << a << " " << #b << " = " << b << endl;

using namespace std;

struct profiler
{
	std::string name;
	std::chrono::high_resolution_clock::time_point p;
	profiler(std::string const &n) : name(n), p(std::chrono::high_resolution_clock::now()) { }

	~profiler() {
		using dura = std::chrono::duration<double>;
		auto d = std::chrono::high_resolution_clock::now() - p;
		std::cout << name << ": " << std::chrono::duration_cast<dura>(d).count() << std::endl;
	}
};

#define PROFILE_BLOCK(pbn) profiler _pfinstance(pbn)

int main123() {
	string filename = "rostro.jpg";
	int scale = 10;

	do {
		cout << "PROGRAMA PARA INTERPOLAR UNA IMAGEN" << endl;
		cout << "===================================" << endl;
		cout << "1.- Interpolar" << endl;
		cout << "2.- convoluci�n" << endl;
		cout << "3.- test convoluci�n CPU" << endl;
		cout << "4.- test convoluci�n GPU" << endl;
		cout << "5.- Salir" << endl;
		
		int switch_on;
		cin >> switch_on;

		switch (switch_on) {
			case 1:  {
					cout << "indique el nivel de zoom (2 - 40):" << endl;
					cin >> scale;
					cout << "el scale ingresado es:" << scale << endl;
					interpolacion* interpolar  = new interpolacion(filename, scale);
					interpolar->vecinos_cercanos();
					interpolar->bilinear();
					interpolar->bicubica();
					//cv::namedWindow("mi ventana para mostrar imagenes", cv::WINDOW_AUTOSIZE);
					cv::imshow("Original", interpolar->get_img());
					cv::imshow("Int.Vecinos", interpolar->get_bigger_img_vecinos());
					cv::imshow("Int.bilineal", interpolar->get_bigger_img_bilineal());
					cv::imshow("Int.bicubica", interpolar->get_bigger_img_bicubica());
					cv::waitKey(0);
					delete interpolar;
					break;
				}
			case 2: {
					convolucion* c = new convolucion("lena_grises.bmp", 3);
					c->deteccion_de_bordes();
					cv::imshow("Original", c->get_original());
					cv::imshow("Con Detecci�n de bordes", c->get_output());
					cv::waitKey(0);
					delete c;
					break;
				}
			case 3: {
				for (int input_size = 0; input_size < 4; input_size++) {
					auto input_wh = 4000 + input_size * 2000;
					for (int mask_size = 1; mask_size <= 7; mask_size++) {
						auto mask_wh = 2 * mask_size + 1;
						for (int num_threads = 1; num_threads < 24; num_threads++) {
							std::cout << "start : " << mask_size << std::endl;
							auto start = std::chrono::high_resolution_clock::now();
							// rows, cols, num_threads
							convolucion c(input_wh, input_wh, mask_wh);
							c.run(num_threads);
							auto diff = std::chrono::high_resolution_clock::now() - start;
							using dura = std::chrono::duration<double>;
							auto tiempo = std::chrono::duration_cast<dura>(diff).count();
							std::cout << "end time" << ": " << tiempo << std::endl;
							std::cout << "para MATRIX_INPUT = " << input_wh << ", para MASK = " << mask_wh << " y NUM_THREADS = " << num_threads << endl;
							std::cout << "============================================================" << endl;
							ofstream out("D:\\temp\\__to_plot_" + std::to_string(input_wh) + "x" + std::to_string(input_wh) + ".txt", ios::app);
							out << mask_wh << "\t" << num_threads << "\t" << tiempo << endl;
							out.close();
						}
						cout << "______________________________________________________________________" << endl;
						cout << "______________________________________________________________________" << endl;
					}
				}
				/*puts("ANTES:");
				c.show_matrices();
				c.run(3);
				puts("DESPUES:");
				c.show_matrices();*/
				break;
			}
			default: break;
		}
		if (switch_on == 4) break;

	} while (true);
	
	return 0;
}

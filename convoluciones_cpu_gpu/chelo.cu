
#include <hip/hip_runtime.h>
//#include "opencv2/highgui/highgui.hpp"
//#include <cstdio>
//#include <time.h>
//#include <sstream>
//#include <iostream>
//
//using namespace cv;
//using namespace std;
//
//#define MAX_THREADS_BY_BLOCK 1024
//#define DIM_BLOCK_X 32
//#define DIM_BLOCK_Y 32
//
//__device__ int cuGPos(int y, int x, int cuCols) {
//	return y * cuCols + x;
//}
//
//__global__ void cudaConvolutionImage(int *cuPoRows, int *cuPoCols, int *cuInImage, int *cuResImage, int* sizeKer, float* kernel){
//	int threadIdGlob = threadIdx.x + blockIdx.x * blockDim.x;
//
//	int cuRows = *cuPoRows;
//	int cuCols = *cuPoCols;
//
//	if (threadIdGlob < cuRows * cuCols) {
//		int y = threadIdGlob / cuCols;
//		int x = threadIdGlob % cuCols;
//		float pixel = 0;
//		for (int i = (*sizeKer) / -2; i <= (*sizeKer) / 2; i++) {
//			for (int j = (*sizeKer) / -2; j <= (*sizeKer) / 2; j++) {
//				if (y + j >= 0 && x + i >= 0) {
//					pixel += cuInImage[cuGPos(y + j, x + i, cuCols)] * kernel[(j + (*sizeKer / 2)*(*sizeKer) + (i + (*sizeKer / 2)))];
//				}
//			}
//		}
//
//		pixel = pixel > 255 ? 255 : pixel;
//		pixel = pixel < 0 ? 0 : pixel;
//		cuResImage[cuGPos(y, x, cuCols)] = pixel;
//		if (cuResImage[cuGPos(y, x, cuCols)] == 0 && x == 0 && y == 0)
//			printf("es igual de cero\n");
//	}
//}
//
//Mat generateConvolutionCUDAGrayImage(Mat inMatIn, float** kernel, int siKe, string nameFile) {
//	Mat inMatImage = inMatIn.clone();
//	float time = 0.0;
//	int nRows = inMatImage.rows;
//	int nCols = inMatImage.cols;
//
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//
//	int* inImage = new int[nRows * nCols];
//	float* kerArray = new float[siKe * siKe];
//
//	for (int x = 0; x < nCols; x++)
//		for (int y = 0; y < nRows; y++)
//			inImage[y*nCols + x] = 0;
//
//	// split in RedGreenBlue channels
//	for (int y = 0; y < nRows; y++){
//		for (int x = 0; x < nCols; x++){
//			inImage[y * nCols + x] = inMatImage.at<uchar>(y, x);
//		}
//	}
//
//	//gen array1d of kernel
//	for (int i = 0; i < siKe; i++)
//		for (int j = 0; j< siKe; j++){
//			kerArray[i*siKe + j] = kernel[i][j];
//		}
//
//	int *cuPoRows, *cuPoCols, *cuN, *cuInImage, *cuResImage;
//	float* cuKernel;
//
//
//	cudaEventRecord(start, 0);
//	cudaMalloc((void**)&cuPoRows, sizeof(int));
//	cudaMalloc((void**)&cuPoCols, sizeof(int));
//	cudaMalloc((void**)&cuN, sizeof(int));
//	cudaMalloc((void**)&cuKernel, siKe * siKe * sizeof(float));
//	cudaMalloc((void**)&cuInImage, nCols * nRows * sizeof(int));
//	cudaMalloc((void**)&cuResImage, nCols * nRows * sizeof(int));
//
//	cudaMemcpy(cuPoRows, &nRows, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuPoCols, &nCols, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuN, &siKe, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuKernel, kerArray, siKe * siKe * sizeof(float), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuInImage, inImage, nCols * nRows * sizeof(int), cudaMemcpyHostToDevice);
//
//	int N = nRows * nCols;
//	dim3 blockDim(MAX_THREADS_BY_BLOCK, 1, 1);
//	dim3 gridDim((N + MAX_THREADS_BY_BLOCK - 1) / MAX_THREADS_BY_BLOCK, 1, 1);
//
//	cudaConvolutionImage << <gridDim, blockDim >> >(cuPoRows, cuPoCols, cuInImage, cuResImage, cuN, cuKernel);
//
//	cudaMemcpy(inImage, cuResImage, nRows * nCols * sizeof(int), cudaMemcpyDeviceToHost);
//
//	cudaFree(cuPoRows);
//	cudaFree(cuPoCols);
//	cudaFree(cuN);
//	cudaFree(cuKernel);
//	cudaFree(cuInImage);
//	cudaFree(cuResImage);
//
//	cudaEventRecord(stop, 0);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&time, start, stop);
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	for (int y = 0; y < nRows; y++)
//		for (int x = 0; x < nCols; x++)
//			inMatImage.at<uchar>(y, x) = inImage[y*nCols + x];
//
//	printf("%f \t", time / 1000.0);
//	return inMatImage;
//}
//
//Mat generateConvolutionCUDARGBImage(Mat inMatImage, float** kernel, int siKe, string nameFile){
//	float time = 0.0;
//	int nRows = inMatImage.rows;
//	int nCols = inMatImage.cols;
//
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//
//	int** inImage = new int*[3];
//	float* kerArray = new float[siKe * siKe];
//	for (int i = 0; i < 3; i++)
//		inImage[i] = new int[nRows * nCols];
//
//	Mat bgr[3];
//	split(inMatImage, bgr);
//
//	// split in RedGreenBlue channels
//	for (int y = 0; y < nRows; y++){
//		for (int x = 0; x < nCols; x++){
//			inImage[0][y * nCols + x] = bgr[0].at<uchar>(y, x);
//			inImage[1][y * nCols + x] = bgr[1].at<uchar>(y, x);
//			inImage[2][y * nCols + x] = bgr[2].at<uchar>(y, x);
//		}
//	}
//
//	//gen array1d of kernel
//	for (int i = 0; i < siKe; i++)
//		for (int j = 0; j< siKe; j++)
//			kerArray[i*siKe + j] = kernel[i][j];
//
//	int *cuPoRows, *cuPoCols, *cuN;
//	int *cuInImageR, *cuInImageG, *cuInImageB;
//	int *cuResImageR, *cuResImageG, *cuResImageB;
//	float *cuKernel;
//
//	cudaEventRecord(start, 0);
//	cudaMalloc((void**)&cuPoRows, sizeof(int));
//	cudaMalloc((void**)&cuPoCols, sizeof(int));
//	cudaMalloc((void**)&cuN, sizeof(int));
//	cudaMalloc((void**)&cuKernel, siKe * siKe * sizeof(float));
//	cudaMalloc((void**)&cuInImageR, nCols * nRows * sizeof(int));
//	cudaMalloc((void**)&cuInImageG, nCols * nRows * sizeof(int));
//	cudaMalloc((void**)&cuInImageB, nCols * nRows * sizeof(int));
//	cudaMalloc((void**)&cuResImageR, nCols * nRows * sizeof(int));
//	cudaMalloc((void**)&cuResImageG, nCols * nRows * sizeof(int));
//	cudaMalloc((void**)&cuResImageB, nCols * nRows * sizeof(int));
//
//	cudaMemcpy(cuPoRows, &nRows, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuPoCols, &nCols, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuN, &siKe, sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuKernel, kerArray, siKe * siKe * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuInImageB, inImage[0], nCols * nRows * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuInImageG, inImage[1], nCols * nRows * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(cuInImageR, inImage[2], nCols * nRows * sizeof(int), cudaMemcpyHostToDevice);
//
//	int N = nRows * nCols;
//	int nBloq = (N + MAX_THREADS_BY_BLOCK - 1) / MAX_THREADS_BY_BLOCK;
//
//	dim3 blockDim(DIM_BLOCK_X, DIM_BLOCK_Y, 1);
//	dim3 gridDim(nBloq, 1, 1);
//
//	cudaConvolutionImage << <gridDim, blockDim >> >(cuPoRows, cuPoCols, cuInImageB, cuResImageB, cuN, cuKernel);
//	cudaConvolutionImage << <gridDim, blockDim >> >(cuPoRows, cuPoCols, cuInImageG, cuResImageG, cuN, cuKernel);
//	cudaConvolutionImage << <gridDim, blockDim >> >(cuPoRows, cuPoCols, cuInImageR, cuResImageR, cuN, cuKernel);
//
//	cudaMemcpy(cuInImageB, inImage[0], nRows * nCols * sizeof(int), cudaMemcpyDeviceToHost);
//	cudaMemcpy(cuInImageG, inImage[1], nRows * nCols * sizeof(int), cudaMemcpyDeviceToHost);
//	cudaMemcpy(cuInImageR, inImage[2], nRows * nCols * sizeof(int), cudaMemcpyDeviceToHost);
//
//	cudaFree(cuPoRows);
//	cudaFree(cuPoCols);
//	cudaFree(cuN);
//	cudaFree(cuKernel);
//	cudaFree(cuInImageB);
//	cudaFree(cuInImageG);
//	cudaFree(cuInImageR);
//	cudaFree(cuResImageB);
//	cudaFree(cuResImageG);
//	cudaFree(cuResImageR);
//
//	cudaEventRecord(stop, 0);
//	cudaEventSynchronize(stop);
//	cudaEventElapsedTime(&time, start, stop);
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//
//	for (int y = 0; y < nRows; y++)
//		for (int x = 0; x < nCols; x++){
//			bgr[0].at<uchar>(y, x) = inImage[0][y*nCols + x];
//			bgr[1].at<uchar>(y, x) = inImage[0][y*nCols + x];
//			bgr[2].at<uchar>(y, x) = inImage[0][y*nCols + x];
//		}
//	merge(bgr, 3, inMatImage);
//	printf("Convoluci�n de CUDA RGB Image:%s Mascara: %d  Tiempo %f\n", nameFile.c_str(), siKe, time / 1000.0);
//	return inMatImage;
//}
//
//string intToString(int n){
//	ostringstream ss;
//	ss << n;
//	string a = n <= 9 ? "0" : "";
//	return a + ss.str();
//}
//
//
//int main111() {
//	string path = "D://temp//convolution//";
//	string imagens[15] = { "gordo.jpg"};
//	string dirOut[15] = { "tekken"};
//
//	//	string imagens[2] = {"wallPaper08.jpeg"};
//	//	string dirOut[1] = {"wallPaper08"};
//
//	int arrayKernel[13] = { 3, 5, 9, 13, 19, 25, 31, 39, 47, 57, 67, 75, 85 };
//	int nImages = 15, nKernels = 13;
//
//	for (int i = 0; i < nImages; i++) {
//		Mat inMatGrayImage = imread(path + imagens[i], CV_LOAD_IMAGE_GRAYSCALE);
//		printf("%s,%dx%d,%ld \n", imagens[i].c_str(), inMatGrayImage.rows, inMatGrayImage.cols, (long)inMatGrayImage.cols * (long)inMatGrayImage.rows);
//		imwrite(path + dirOut[i] + "/grayScale" + imagens[i], inMatGrayImage);
//		for (int j = 0; j < nKernels; j++) {
//			float **kernel = new float*[arrayKernel[j]];
//			for (int k = 0; k < arrayKernel[j]; k++) {
//				kernel[k] = new float[arrayKernel[j]];
//				for (int l = 0; l < arrayKernel[j]; l++)
//					kernel[k][l] = 1.0 / (arrayKernel[j] * arrayKernel[j]);
//			}
//			Mat outMatImage = generateConvolutionCUDAGrayImage(inMatGrayImage, kernel, arrayKernel[j], imagens[i]);
//			string fullpath = path + dirOut[i] + "//grayCuda_" + intToString(arrayKernel[j]) + "_" + imagens[i];
//			cout << "kernel : " << arrayKernel[j] << " - fullpath : " << fullpath << endl;
//			imwrite(fullpath, outMatImage);
//		}
//		printf("\n");
//	}
//
//	return 0;
//}

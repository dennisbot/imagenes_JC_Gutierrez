#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <sstream>
#include <algorithm>
#include "Bosel.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <sys/stat.h>

//#define cimg_use_jpeg
#include "CImg.h"

#define SIZE 10

#define db(a) cout << #a << " = " << a << endl
#define db2(a, b) cout << #a << " = " << a << " " << #b << " = " << b << endl

inline bool existe(const std::string& name) {
	struct stat buffer;
	return (stat(name.c_str(), &buffer) == 0);
}

void performCPU(string filename)
{
	std::clock_t start = clock();
	double duration;
	if (existe(filename)) {
		puts("EL ARCHIVO SI EXISTE");
	}
	else {
		puts("EL ARCHIVO NO EXISTE");
	}
	ImgFloat imagen(filename.c_str());
	//ImgFloat imagen("lena30.jpg");
	// depth, numColors, initialize
	ImgFloat xGradient(imagen.width(), imagen.height(), 1, 1, 0);
	ImgFloat yGradient(imagen.width(), imagen.height(), 1, 1, 0);
	ImgFloat gradientA(imagen.width(), imagen.height(), 1, 1, 0);
	ImgFloat gradientB(imagen.width(), imagen.height(), 1, 1, 0);

	//imagen.blur(1.5);

	ImgFloat R = imagen.get_channel(0);

	Bosel b;
	b.convolution(R, b.Gx, xGradient);
	b.convolution(R, b.Gy, yGradient);

	b.mergeA(gradientA, xGradient, yGradient);
	b.mergeB(gradientB, xGradient, yGradient);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	printf("CPU for image %s takes %.2f seconds\n", filename.c_str(), duration);

	(gradientA, gradientB).display("comparaci�n suma ABSs y SQRT");
	//(xGradient, yGradient, gradient).display("Detecci�n de Bordes");
	cout << duration << endl;
	//gradient.display();
}

__device__ void convolution(int coordinate, float* d_arr, float* gradient, int width, int len, int* mask, int* dir, int* pos)
{
	float c = 0;
	for (int ii = 0; ii < 3; ii++)
	{
		for (int jj = 0; jj < 3; jj++)
		{
			int x = coordinate + width * dir[ii * 3 + jj] + pos[jj];
			if (x >= 0 && x < len)
				c += d_arr[x] * mask[ii * 3 + jj];
		}
	}
	gradient[coordinate] = c;
}

__global__ void deviceComputeGradient(float* d_arr, float* gradient, int width, int len, int* mask, int* dir, int* pos) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x * width + y < len)
		convolution(x * width + y, d_arr, gradient, width, len, mask, dir, pos);
}

__global__ void deviceMerge(float* xGradient, float* yGradient, float* target, int width, int len) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int index = x * width + y;
	if (index < len)
		target[index] = abs(xGradient[index]) + abs(yGradient[index]);
}

//segunda versi�n (un s�lo m�todo)
__device__ void convolution2(int coordinate, float* d_arr, float* gradient, int width, int len, int* gx_mask, int* gy_mask, int* dir, int* pos)
{
	float cx = 0, cy = 0;
	for (int ii = 0; ii < 3; ii++)
	{
		for (int jj = 0; jj < 3; jj++)
		{
			int x = coordinate + width * dir[ii * 3 + jj] + pos[jj];
			if (x >= 0 && x < len) {
				cx += d_arr[x] * gx_mask[ii * 3 + jj];
				cy += d_arr[x] * gy_mask[ii * 3 + jj];
			}
		}
	}
	gradient[coordinate] = abs(cx) + abs(cy);
}

__global__ void deviceComputeGradient2(float* d_arr, float* gradient, int width, int len, int* gx_mask, int* gy_mask, int* dir, int* pos) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x == 0 && y < 10)
		printf("[%d %d]\n", x, y);
	if (x * width + y < len)
		convolution2(x * width + y, d_arr, gradient, width, len, gx_mask, gy_mask, dir, pos);
}


void performGPU(string filename)
{
	ImgFloat imagen(filename.c_str());

	std::clock_t startt = clock();
	double duration;
	ImgFloat result(imagen.width(), imagen.height(), 1, 1, 0);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// depth, numColors, initialize
	int WIDTH = imagen.width();
	int HEIGHT = imagen.height();
	float *arr, *gradient;
	float *d_arr, *d_gradient;

	arr = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
	gradient = (float*)malloc(WIDTH * HEIGHT * sizeof(float));

	hipMalloc((void**)&d_arr, WIDTH * HEIGHT * sizeof(float));
	hipMalloc((void**)&d_gradient, WIDTH * HEIGHT * sizeof(float));

	for (int i = 0; i < WIDTH; i++)
		for (int j = 0; j < HEIGHT; j++) {
			arr[i * WIDTH + j] = imagen(i, j);
			gradient[i * WIDTH + j] = 0;
		}

	hipMemcpy(d_arr, arr, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_gradient, gradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyHostToDevice);

	dim3 BLOCKS = dim3(1024, 1024);
	dim3 THREADS = dim3(4, 4);

	/*dim3 BLOCKS(2, 2);
	dim3 THREADS(2, 2);*/

	int pos[3] = { -1, 0, 1 };
	int dir[9] = { -1, -1, -1, 0, 0, 0, 1, 1, 1 };
	int Gx[9] = {
		-1, 0, 1,
		-2, 0, 2,
		-1, 0, 1
	};
	int Gy[9] = {
		1, 2, 1,
		0, 0, 0,
		-1, -2, -1
	};
	int* d_pos, *d_dir, *d_Gx, *d_Gy;

	hipMalloc((void**)&d_pos, 3 * sizeof(int));
	hipMalloc((void**)&d_dir, 9 * sizeof(int));
	hipMalloc((void**)&d_Gx, 9 * sizeof(int));
	hipMalloc((void**)&d_Gy, 9 * sizeof(int));

	hipMemcpy(d_pos, pos, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dir, dir, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Gx, Gx, 9 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Gy, Gy, 9 * sizeof(float), hipMemcpyHostToDevice);


	hipEventRecord(start);
	deviceComputeGradient2 << < BLOCKS, THREADS >> > (d_arr, d_gradient, WIDTH, WIDTH * HEIGHT, d_Gx, d_Gy, d_dir, d_pos);
	hipMemcpy(gradient, d_gradient, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(stop);

	hipEventSynchronize(stop);

	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < HEIGHT; j++) {
			result(i, j) = gradient[i * WIDTH + j];
		}
	}


	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	duration = (std::clock() - startt) / (double)CLOCKS_PER_SEC;

	printf("GPU for image %s takes %.2f seconds\n", filename.c_str(), duration);
	printf("hipEventElapsedTime = %.2f ms\n", milliseconds);

	free(arr);


	hipFree(d_arr);
	hipFree(d_gradient);

	hipFree(d_pos);
	hipFree(d_dir);
	hipFree(d_Gx);
	hipFree(d_Gy);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	//ac� si quieres ponemos en xgradient, ygradient
	(result).display("HOLA MUNDO CUDA");
}


int main(int argc, char** argv) {
	CImg<unsigned char> image("lena_grises.bmp"), visu(500, 400, 1, 3, 0);
	const unsigned char red[] = { 255, 0, 0 }, green[] = { 0, 255, 0 }, blue[] = { 0, 0, 255 };
	image.blur(2.5);
	CImgDisplay main_disp(image, "Click a point"), draw_disp(visu, "Intensity profile");
	while (!main_disp.is_closed() && !draw_disp.is_closed()) {
		main_disp.wait();
		if (main_disp.button() && main_disp.mouse_y() >= 0) {
			const int y = main_disp.mouse_y();
			visu.fill(0).draw_graph(image.get_crop(0, y, 0, 0, image.width() - 1, y, 0, 0), red, 1, 1, 0, 255, 0);
			visu.draw_graph(image.get_crop(0, y, 0, 1, image.width() - 1, y, 0, 1), green, 1, 1, 0, 255, 0);
			visu.draw_graph(image.get_crop(0, y, 0, 2, image.width() - 1, y, 0, 2), blue, 1, 1, 0, 255, 0).display(draw_disp);
		}
	}
	return 0;


	performCPU("lena.jpg");
	return 0;
	for (int i = 1; i < 2; i++)
	{
		int len = 4;
		int baseSize = 1024;
		for (int j = 1; j <= len; j++)
		{
			ostringstream stream;
			stream << (baseSize * j);
			if (i == 0)
				performCPU(stream.str() + "x" + stream.str() + ".jpg");
			else
				performGPU(stream.str() + "x" + stream.str() + ".jpg");
		}
	}

	return 0;
}
